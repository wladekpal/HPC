#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>
#include <fstream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>

#include "clicque.cu"


int main(int argc, char** argv)
{
    if (argc != 4) {
        std::cout << "Usage ./kcliques graph_input k output" << std::endl;
        return 1;
    }

    std::string input_file_name = std::string(argv[1]);
    int k = std::stoi(argv[2]);
    std::string output_file_name = std::string(argv[3]);

    std::ifstream input_file(input_file_name);

    std::vector<std::pair<int,int>> v;

    int x, y, vertex_count;
    while(input_file >> x >> y) {
        // std::cout << "Siema, " << x << y << std::endl;
        vertex_count = std::max(vertex_count, x);
        vertex_count = std::max(vertex_count, y);
        v.push_back(std::make_pair(x, y));
    }

    std::cout << "vertex_count " << vertex_count << std::endl;

    thrust::host_vector<thrust::host_vector<int>> edges(vertex_count);

    for(int i = 0; i < v.size(); i++) {
        edges[v[i].first].push_back(v[i].second);
    }

    thrust::host_vector<int> result = find_cliques(edges, k);

    std::ofstream output_file(output_file_name);

    for(int i = 0; i < result.size(); i++){
        output_file << result[i] << " ";
    }

    return 0;
}