#include "hip/hip_runtime.h"
#include <cstdio>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/set_operations.h>

static void handleError(hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

#define TILE_DIM 32
#define BLOCK_ROWS 8

#define MAX_STACK_SIZE 12
#define MAX_DEGREE 1024
#define GROUP_SIZE 32
#define BIN_ENC_SIZE 32
#define FULL_32_FLAG 0xffffffff


// Kernel for induced sub_graph extraction
__global__ void induced_subgraph_extraction_kernel(
        int *edges, // SORTED!!!!!
        int *edge_sizes,
        unsigned int *result,
        int num_v) {
    
    int start_vertex = blockIdx.x; // TODO probably should be in for loop, there can be more verticies in graph than blocks
    // int group_id = threadIdx.x;
    int group_size = blockDim.x;
    int thread_id = threadIdx.x;


    //Here extract induced subgraph 
    __shared__ unsigned int start_edges[MAX_DEGREE];

    for (int i = thread_id; i < edge_sizes[start_vertex]; i++) {
        start_edges[i] = edges[start_vertex * MAX_DEGREE + i]; // TODO: czy tutaj MAX_DEGREE ???
    }

    for (int i = thread_id; i < edge_sizes[start_vertex]; i+= group_size) {
        unsigned int enc = 0;
        unsigned int factor = 0;
        unsigned int counter_neighbour = 0;
        unsigned int counter_start = 0;
        unsigned int current_vertex = edges[start_vertex * MAX_DEGREE + i];

        while(counter_start < edge_sizes[start_vertex] && counter_neighbour < edge_sizes[current_vertex]) {
            if (start_edges[counter_start] == edges[current_vertex * MAX_DEGREE + counter_neighbour]) {
                enc |= 1;
                counter_start++;
                counter_neighbour++;
                enc = enc << 1;
            }
            else if (start_edges[counter_start] < edges[current_vertex * MAX_DEGREE + counter_neighbour]) {
                counter_start++;
                enc = enc << 1;
            }
            else {
                counter_neighbour++;
            }

            if (counter_start % BIN_ENC_SIZE == 31) {
                result[start_vertex * num_v * MAX_DEGREE + i * MAX_DEGREE + factor] = enc;
                enc = 0;
                factor++;
            }
        }

        if (counter_start % BIN_ENC_SIZE != 31) {
                result[start_vertex * num_v * MAX_DEGREE + i * MAX_DEGREE + factor] = enc << (31 - counter_start); //TODO: tutaj chyba jakoś inaczej
                enc = 0;
                factor++;
        }
    }
}

__device__ int get_next_vertex_idx(
        unsigned int *list, // int_32[(MAX_DEGREE / BIN_ENC_SIZE)]
        int *idx, // index of group in list above (from right side)
        unsigned int *mask_idx // Index inside of group (from right side)
    ) {

    while (*idx < (MAX_DEGREE / BIN_ENC_SIZE)) {
        unsigned int x = list[*idx] & (FULL_32_FLAG << *mask_idx);
        x = __ffs(x);

        if (x != 0){
            *mask_idx = x ;
            x = 32 - x;

            if (*mask_idx >= 31) {
                *mask_idx = 0;
                (*idx)++;
            }

            return x;
        }

        (*idx)++;
        *mask_idx = 0;
    }

    return -1;
}


__device__ unsigned int get_number_of_bits(
        unsigned int *list ) {
    unsigned int result = 0;
    for (int i = 0; i < (MAX_DEGREE / BIN_ENC_SIZE); i++) {
        result += __popc(list[i]);
    }

    return result;
}


// Kernel of vertex-centric graph orientation approach                                     
__global__ void graph_orientation_kernel(
        unsigned int *sub_graphs, // int_32[NUM_V][MAX_DEGREE][(MAX_DEGREE / BIN_ENC_SIZE)] TODO: maybe it's better to extract it in this function ???
        unsigned int *num_cliques, 
        unsigned int k,
        unsigned int num_v) {

    // Stack in SHARED memory (pre-allocated)
    // Counters on stack in SHARED memory
    // Vertex lists on stack in GLOBAL memory
    
    int start_vertex = blockIdx.x; // TODO probably should be in for loop, there can be more verticies in graph than blocks
    int group_id = blockIdx.y;
    int group_size = blockDim.x;
    int thread_id = threadIdx.x;

    int verticies[MAX_STACK_SIZE]; // Currently worked vertex index in adjacency list from level above for each recursion depth
    unsigned int masks[MAX_STACK_SIZE];
    unsigned int current_adj_size[MAX_STACK_SIZE];
    unsigned int current_level; // Level of recursion


    __shared__ unsigned int sub_graph_adj[MAX_STACK_SIZE][(MAX_DEGREE / BIN_ENC_SIZE)];
    // Initialization (first depth of recursion)
    current_level = 1;
    verticies[0] = 0;

    for (int i = thread_id; i < (MAX_DEGREE / BIN_ENC_SIZE); i += group_size) {
        sub_graph_adj[0][i] = sub_graphs[start_vertex * num_v * MAX_DEGREE + group_id * MAX_DEGREE + i];
    }
    verticies[0] = 0;
    masks[0] = 0;

    while(true) {
        // Get info about current task
        // int current_vertex_idx = verticies[current_level]; // Index of current vertex on adjacency list from level above
        // unsigned int prev_size = current_adj_size[current_level - 1]; //Size of adjacency list on higher level
        
        // Update verticies
        if (thread_id == 0){
            printf("KERNEL: group: %d id: %d verticies %d masks %d graph: %u\n", group_id, thread_id, verticies[current_level-1], masks[current_level-1], sub_graph_adj[current_level-1][0]);
        }
        int current_vertex_idx = get_next_vertex_idx(sub_graph_adj[current_level-1], &(verticies[current_level-1]), &(masks[current_level-1]));

        if (thread_id == 0){
            printf("KERNEL: group: %d id: %d vertex_idx %d level: %d\n", group_id, thread_id, current_vertex_idx, current_level);
        }
        // If current vertex is beyond the adjacency list from previous level, we have exhausted a level,
        // and need to backtrack.
        if (current_vertex_idx < 0) {
            if (current_level == 0)
                return;

            current_level--;
            

            continue;
        }

        for (int i = thread_id; i < (MAX_DEGREE / BIN_ENC_SIZE); i+= group_size) {
            sub_graph_adj[current_level][i] = sub_graph_adj[current_level-1][i] & sub_graphs[start_vertex * num_v * MAX_DEGREE + current_vertex_idx * MAX_DEGREE + i];
        }

        if (thread_id == 0) {
            printf("KERNEL: group: %d id: %d adjacency list: \n", group_id, thread_id);
            for (int i = 0; i < 2; i++) {
                printf("%u %u %u \n",sub_graph_adj[current_level-1][i], sub_graphs[start_vertex * num_v * MAX_DEGREE + current_vertex_idx * MAX_DEGREE + i], sub_graph_adj[current_level][i]);
            }

            printf("\n");
        }

        // Vertex that the group is working on in current iteration.
        // int current_vertex = group_adj_lists[current_level - 1][current_vertex_idx]; // Vertex id in GRAPH



        unsigned int current_size = get_number_of_bits(sub_graph_adj[current_level]);

        if (thread_id == 0) {
            printf("KERNEL: group: %d id: %d current_size: %u \n", group_id, thread_id, current_size);
        }

       
        if (current_level + 3 == k) { // If we are at k-th level of recursion we have to accumulate an answer and backtrack

            if (thread_id == 0) {
                printf("Clicque found, %u\n", current_size);
                //atomicAdd(num_cliques, current_size);
                
            }

            // Go one level above in recursion
            // current_level--;

            
            // If we have reached the top of recursion it means that we have searched through the whole subgraph

        }
        else if (current_size > 0) { // If there is still something in adjacency list we go deeper

            // Go one level deeper in recursion
            current_level++;
            // Mark that we are starting from the beginnig on this level of recursion
            verticies[current_level] = 0;
            masks[current_level] = 0;
        }
        
    }
}


// Vertex-centric graph orientation approach
thrust::host_vector<int> graph_orientation(thrust::host_vector<thrust::host_vector<int>> edges, int k) {
    std::cout << "GRAPH ORIENTATION BEGININNG" << std::endl;
    int *dev_edges;
    int *dev_sizes;
    unsigned int *dev_num_cliques;
    unsigned int *dev_results;

    int host_edges[edges.size() * MAX_DEGREE];
    int host_sizes[edges.size()];
    unsigned int host_results[edges.size() * MAX_DEGREE * (MAX_DEGREE / BIN_ENC_SIZE)];
    
    hipMalloc((void**)&dev_edges, sizeof(int) * edges.size() * MAX_DEGREE);
    hipMalloc((void**)&dev_sizes, sizeof(int) * edges.size());
    hipMalloc((void**)&dev_results, sizeof(unsigned int) * edges.size() * MAX_DEGREE * (MAX_DEGREE / BIN_ENC_SIZE));
    hipMalloc((void**)&dev_num_cliques, sizeof(int));

    for (int i = 0; i < edges.size();i++) {
        std::cout << i << ": ";
        for (int j = 0; j < edges[i].size(); j++) {
            std::cout << edges[i][j] << " ";
            host_edges[i * MAX_DEGREE + j] = edges[i][j];
            
        }
        std::cout << " -> " << edges[i].size() << std::endl;
        host_sizes[i] = edges[i].size();
    }

    std::cout << "data copied to host" << std::endl;

    unsigned int num_cliques = 0;

    hipMemcpy(dev_edges, host_edges, sizeof(int) * edges.size() * MAX_DEGREE, hipMemcpyHostToDevice);
    hipMemcpy(dev_sizes, host_sizes, sizeof(int) * edges.size(), hipMemcpyHostToDevice);
    hipMemcpy(dev_num_cliques, &num_cliques, sizeof(int), hipMemcpyHostToDevice);

    std::cout << "Everything copied to device, starting kernel" << std::endl;

    
    induced_subgraph_extraction_kernel<<<edges.size(),GROUP_SIZE>>>(dev_edges, dev_sizes, dev_results, edges.size());
    cudaCheck(hipPeekAtLastError());

    hipDeviceSynchronize();

    hipMemcpy(host_results, dev_results, edges.size() * MAX_DEGREE * (MAX_DEGREE / BIN_ENC_SIZE), hipMemcpyDeviceToHost);

    std::cout << "Results for " <<std::endl;
    for (int l = 0; l < edges.size();l++){
        std::cout << "vertex " << l << std::endl;
        for (int i = 0; i < edges.size(); i++) {
            for (int j = 0; j < 2; j++) {
                std::cout << host_results[l * edges.size() * MAX_DEGREE + i * MAX_DEGREE + j] << " ";
            }
            std::cout << std::endl;
        }
        std::cout << std::endl;
    }
    printf("\n\n\n\n\n");

    
    // dim3 dimGrid(edges.size(), MAX_DEGREE, 1);
    dim3 dimGrid(1, 1, 1);
    graph_orientation_kernel<<<dimGrid, GROUP_SIZE>>>(dev_results, &num_cliques, k, edges.size());
    cudaCheck(hipPeekAtLastError());
    hipDeviceSynchronize();

    hipMemcpy(&num_cliques, dev_num_cliques, sizeof(int), hipMemcpyDeviceToHost);
    printf("NUMBER OF CLIQUES %d\n", num_cliques);

    return thrust::host_vector<int>(10, 1);
}


__global__ void test_kernel() {
    int idx = 0;
    unsigned int mask_idx = 0;
    unsigned int list[(MAX_DEGREE / BIN_ENC_SIZE)];

    list[0] = 0b01000000000000000000000000000010;
    list[1] = 0b00000000000000000000010000000000;
    int result;
    result = get_next_vertex_idx(list, &idx, &mask_idx);
    printf("Result %d\n", result);

    result = get_next_vertex_idx(list, &idx, &mask_idx);
    printf("Result %d\n", result);

    result = get_next_vertex_idx(list, &idx, &mask_idx);
    printf("Result %d\n", result);
}

thrust::host_vector<int> find_cliques(thrust::host_vector<thrust::host_vector<int>> edges, int k) {

    // test_kernel<<<1,1>>>();
    // hipDeviceSynchronize();
    thrust::host_vector<int> result = graph_orientation(edges, k);


    return thrust::host_vector<int>(10, 1);
}